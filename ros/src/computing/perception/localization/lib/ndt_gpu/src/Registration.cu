#include "hip/hip_runtime.h"
/*
 * Copyright 2015-2019 Autoware Foundation. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
/*
 * Software License Agreement (BSD License)
 *
 *  Point Cloud Library (PCL) - www.pointclouds.org
 *  Copyright (c) 2010-2011, Willow Garage, Inc.
 *  Copyright (c) 2012-, Open Perception, Inc.
 *
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of the copyright holder(s) nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 */

#include "ndt_gpu/Registration.h"
#include "ndt_gpu/debug.h"
#include <iostream>

namespace gpu {

GRegistration::GRegistration()
{
	max_iterations_ = 0;
	x_ = y_ = z_ = NULL;
	points_number_ = 0;

	trans_x_ = trans_y_ = trans_z_ = NULL;

	converged_ = false;
	nr_iterations_ = 0;

	transformation_epsilon_ = 0;
	target_cloud_updated_ = true;
	target_points_number_ = 0;

	target_x_ = target_y_ = target_z_ = NULL;
	is_copied_ = false;

}

GRegistration::GRegistration(const GRegistration &other)
{
	transformation_epsilon_ = other.transformation_epsilon_;
	max_iterations_ = other.max_iterations_;

	//Original scanned point clouds
	x_ = other.x_;
	y_ = other.y_;
	z_ = other.z_;

	points_number_ = other.points_number_;

	trans_x_ = other.trans_x_;
	trans_y_ = other.trans_y_;
	trans_z_ = other.trans_z_;

	converged_ = other.converged_;

	nr_iterations_ = other.nr_iterations_;
	final_transformation_ = other.final_transformation_;
	transformation_ = other.transformation_;
	previous_transformation_ = other.previous_transformation_;

	target_cloud_updated_ = other.target_cloud_updated_;

	target_x_ = other.target_x_;
	target_y_ = other.target_y_;
	target_z_ = other.target_z_;

	target_points_number_ = other.target_points_number_;
	is_copied_ = true;
}

GRegistration::~GRegistration()
{
	if (!is_copied_) {
		if (x_ != NULL) {
			checkCudaErrors(hipFree(x_));
			x_ = NULL;
		}

		if (y_ != NULL) {
			checkCudaErrors(hipFree(y_));
			y_ = NULL;
		}

		if (z_ != NULL) {
			checkCudaErrors(hipFree(z_));
			z_ = NULL;
		}

		if (trans_x_ != NULL) {
			checkCudaErrors(hipFree(trans_x_));
			trans_x_ = NULL;
		}

		if (trans_y_ != NULL) {
			checkCudaErrors(hipFree(trans_y_));
			trans_y_ = NULL;
		}

		if (trans_z_ != NULL) {
			checkCudaErrors(hipFree(trans_z_));
			trans_z_ = NULL;
		}

		if (target_x_ != NULL) {
				checkCudaErrors(hipFree(target_x_));
			target_x_ = NULL;
		}

		if (target_y_ != NULL) {
			checkCudaErrors(hipFree(target_y_));
			target_y_ = NULL;
		}

		if (target_z_ != NULL) {
			checkCudaErrors(hipFree(target_z_));
			target_z_ = NULL;
		}
	}
}

void GRegistration::setTransformationEpsilon(double trans_eps)
{
	transformation_epsilon_ = trans_eps;
}

double GRegistration::getTransformationEpsilon() const
{
	return transformation_epsilon_;
}

void GRegistration::setMaximumIterations(int max_itr)
{
	max_iterations_ = max_itr;
}

int GRegistration::getMaximumIterations() const
{
	return max_iterations_;
}

Eigen::Matrix<float, 4, 4> GRegistration::getFinalTransformation() const
{
	return final_transformation_;
}

int GRegistration::getFinalNumIteration() const
{
	return nr_iterations_;
}

bool GRegistration::hasConverged() const
{
	return converged_;
}


template <typename T>
__global__ void convertInput(T *input, float *out_x, float *out_y, float *out_z, int point_num)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = idx; i < point_num; i += stride) {
		T tmp = input[i];
		out_x[i] = tmp.x;
		out_y[i] = tmp.y;
		out_z[i] = tmp.z;
	}
}

void GRegistration::setInputSource(pcl::PointCloud<pcl::PointXYZI>::Ptr input)
{
	//Convert point cloud to float x, y, z
	if (input->size() > 0) {
		points_number_ = input->size();

		pcl::PointXYZI *tmp;

		checkCudaErrors(hipMalloc(&tmp, sizeof(pcl::PointXYZI) * points_number_));

		pcl::PointXYZI *host_tmp = input->points.data();

		// Pin the host buffer for accelerating the memory copy
#ifndef __aarch64__
		checkCudaErrors(hipHostRegister(host_tmp, sizeof(pcl::PointXYZI) * points_number_, hipHostRegisterDefault));
#endif

		checkCudaErrors(hipMemcpy(tmp, host_tmp, sizeof(pcl::PointXYZI) * points_number_, hipMemcpyHostToDevice));

		if (x_ != NULL) {
			checkCudaErrors(hipFree(x_));
			x_ = NULL;
		}

		if (y_ != NULL) {
			checkCudaErrors(hipFree(y_));
			y_ = NULL;
		}

		if (z_ != NULL) {
			checkCudaErrors(hipFree(z_));
			z_ = NULL;
		}

		checkCudaErrors(hipMalloc(&x_, sizeof(float) * points_number_));
		checkCudaErrors(hipMalloc(&y_, sizeof(float) * points_number_));
		checkCudaErrors(hipMalloc(&z_, sizeof(float) * points_number_));

		int block_x = (points_number_ > BLOCK_SIZE_X) ? BLOCK_SIZE_X : points_number_;
		int grid_x = (points_number_ - 1) / block_x + 1;

		convertInput<pcl::PointXYZI><<<grid_x, block_x>>>(tmp, x_, y_, z_, points_number_);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());


		if (trans_x_ != NULL) {
			checkCudaErrors(hipFree(trans_x_));
			trans_x_ = NULL;
		}

		if (trans_y_ != NULL) {
			checkCudaErrors(hipFree(trans_y_));
			trans_y_ = NULL;
		}

		if (trans_z_ != NULL) {
			checkCudaErrors(hipFree(trans_z_));
			trans_z_ = NULL;
		}

		checkCudaErrors(hipMalloc(&trans_x_, sizeof(float) * points_number_));
		checkCudaErrors(hipMalloc(&trans_y_, sizeof(float) * points_number_));
		checkCudaErrors(hipMalloc(&trans_z_, sizeof(float) * points_number_));

		// Initially, also copy scanned points to transformed buffers
		checkCudaErrors(hipMemcpy(trans_x_, x_, sizeof(float) * points_number_, hipMemcpyDeviceToDevice));
		checkCudaErrors(hipMemcpy(trans_y_, y_, sizeof(float) * points_number_, hipMemcpyDeviceToDevice));
		checkCudaErrors(hipMemcpy(trans_z_, z_, sizeof(float) * points_number_, hipMemcpyDeviceToDevice));

		checkCudaErrors(hipFree(tmp));

		// Unpin host buffer
#ifndef __aarch64__
		checkCudaErrors(hipHostUnregister(host_tmp));
#endif
	}
}

void GRegistration::setInputSource(pcl::PointCloud<pcl::PointXYZ>::Ptr input)
{
	//Convert point cloud to float x, y, z
	if (input->size() > 0) {
		points_number_ = input->size();

		pcl::PointXYZ *tmp;

		checkCudaErrors(hipMalloc(&tmp, sizeof(pcl::PointXYZ) * points_number_));

		pcl::PointXYZ *host_tmp = input->points.data();

		// Pin the host buffer for accelerating the memory copy
#ifndef __aarch64__
		checkCudaErrors(hipHostRegister(host_tmp, sizeof(pcl::PointXYZ) * points_number_, hipHostRegisterDefault));
#endif

		checkCudaErrors(hipMemcpy(tmp, host_tmp, sizeof(pcl::PointXYZ) * points_number_, hipMemcpyHostToDevice));

		if (x_ != NULL) {
			checkCudaErrors(hipFree(x_));
			x_ = NULL;
		}

		if (y_ != NULL) {
			checkCudaErrors(hipFree(y_));
			y_ = NULL;
		}

		if (z_ != NULL) {
			checkCudaErrors(hipFree(z_));
			z_ = NULL;
		}

		checkCudaErrors(hipMalloc(&x_, sizeof(float) * points_number_));
		checkCudaErrors(hipMalloc(&y_, sizeof(float) * points_number_));
		checkCudaErrors(hipMalloc(&z_, sizeof(float) * points_number_));

		int block_x = (points_number_ > BLOCK_SIZE_X) ? BLOCK_SIZE_X : points_number_;
		int grid_x = (points_number_ - 1) / block_x + 1;

		convertInput<pcl::PointXYZ><<<grid_x, block_x>>>(tmp, x_, y_, z_, points_number_);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());

		if (trans_x_ != NULL) {
			checkCudaErrors(hipFree(trans_x_));
			trans_x_ = NULL;
		}

		if (trans_y_ != NULL) {
			checkCudaErrors(hipFree(trans_y_));
			trans_y_ = NULL;
		}

		if (trans_z_ != NULL) {
			checkCudaErrors(hipFree(trans_z_));
			trans_z_ = NULL;
		}

		checkCudaErrors(hipMalloc(&trans_x_, sizeof(float) * points_number_));
		checkCudaErrors(hipMalloc(&trans_y_, sizeof(float) * points_number_));
		checkCudaErrors(hipMalloc(&trans_z_, sizeof(float) * points_number_));

		checkCudaErrors(hipMemcpy(trans_x_, x_, sizeof(float) * points_number_, hipMemcpyDeviceToDevice));
		checkCudaErrors(hipMemcpy(trans_y_, y_, sizeof(float) * points_number_, hipMemcpyDeviceToDevice));
		checkCudaErrors(hipMemcpy(trans_z_, z_, sizeof(float) * points_number_, hipMemcpyDeviceToDevice));

		checkCudaErrors(hipFree(tmp));
#ifndef __aarch64__
		checkCudaErrors(hipHostUnregister(host_tmp));
#endif
	}
}



//Set input MAP data
void GRegistration::setInputTarget(pcl::PointCloud<pcl::PointXYZI>::Ptr input)
{
	if (input->size() > 0) {
		target_points_number_ = input->size();

		pcl::PointXYZI *tmp;

		checkCudaErrors(hipMalloc(&tmp, sizeof(pcl::PointXYZI) * target_points_number_));

		pcl::PointXYZI *host_tmp = input->points.data();

#ifndef __aarch64__
		checkCudaErrors(hipHostRegister(host_tmp, sizeof(pcl::PointXYZI) * target_points_number_, hipHostRegisterDefault));
#endif

		checkCudaErrors(hipMemcpy(tmp, host_tmp, sizeof(pcl::PointXYZI) * target_points_number_, hipMemcpyHostToDevice));

		if (target_x_ != NULL) {
			checkCudaErrors(hipFree(target_x_));
			target_x_ = NULL;
		}

		if (target_y_ != NULL) {
			checkCudaErrors(hipFree(target_y_));
			target_y_ = NULL;
		}

		if (target_z_ != NULL) {
			checkCudaErrors(hipFree(target_z_));
			target_z_ = NULL;
		}

		checkCudaErrors(hipMalloc(&target_x_, sizeof(float) * target_points_number_));
		checkCudaErrors(hipMalloc(&target_y_, sizeof(float) * target_points_number_));
		checkCudaErrors(hipMalloc(&target_z_, sizeof(float) * target_points_number_));

		int block_x = (target_points_number_ > BLOCK_SIZE_X) ? BLOCK_SIZE_X : target_points_number_;
		int grid_x = (target_points_number_ - 1) / block_x + 1;

		convertInput<pcl::PointXYZI><<<grid_x, block_x>>>(tmp, target_x_, target_y_, target_z_, target_points_number_);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());

#ifndef __aarch64__
		checkCudaErrors(hipHostUnregister(host_tmp));
#endif
		checkCudaErrors(hipFree(tmp));
	}
}

void GRegistration::setInputTarget(pcl::PointCloud<pcl::PointXYZ>::Ptr input)
{
	if (input->size() > 0) {
		target_points_number_ = input->size();

		pcl::PointXYZ *tmp;

		checkCudaErrors(hipMalloc(&tmp, sizeof(pcl::PointXYZ) * target_points_number_));

		pcl::PointXYZ *host_tmp = input->points.data();

#ifndef __aarch64__
		checkCudaErrors(hipHostRegister(host_tmp, sizeof(pcl::PointXYZ) * target_points_number_, hipHostRegisterDefault));
#endif

		checkCudaErrors(hipMemcpy(tmp, host_tmp, sizeof(pcl::PointXYZ) * target_points_number_, hipMemcpyHostToDevice));

		if (target_x_ != NULL) {
			checkCudaErrors(hipFree(target_x_));
			target_x_ = NULL;
		}

		if (target_y_ != NULL) {
			checkCudaErrors(hipFree(target_y_));
			target_y_ = NULL;
		}

		if (target_z_ != NULL) {
			checkCudaErrors(hipFree(target_z_));
			target_z_ = NULL;
		}

		checkCudaErrors(hipMalloc(&target_x_, sizeof(float) * target_points_number_));
		checkCudaErrors(hipMalloc(&target_y_, sizeof(float) * target_points_number_));
		checkCudaErrors(hipMalloc(&target_z_, sizeof(float) * target_points_number_));

		int block_x = (target_points_number_ > BLOCK_SIZE_X) ? BLOCK_SIZE_X : target_points_number_;
		int grid_x = (target_points_number_ - 1) / block_x + 1;

		convertInput<pcl::PointXYZ><<<grid_x, block_x>>>(tmp, target_x_, target_y_, target_z_, target_points_number_);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());

		checkCudaErrors(hipFree(tmp));
#ifndef __aarch64__
		checkCudaErrors(hipHostUnregister(host_tmp));
#endif
	}
}

void GRegistration::align(const Eigen::Matrix<float, 4, 4> &guess)
{
	converged_ = false;

	final_transformation_ = transformation_ = previous_transformation_ = Eigen::Matrix<float, 4, 4>::Identity();

	computeTransformation(guess);
}

void GRegistration::computeTransformation(const Eigen::Matrix<float, 4, 4> &guess) {
	printf("Unsupported by Registration\n");
}

}
